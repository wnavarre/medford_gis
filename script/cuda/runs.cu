#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "runs.h"
#include <iostream>

extern "C" {

namespace {

static constexpr uint8_t StepNone  = 0;
static constexpr uint8_t StepRight = 1;
static constexpr uint8_t StepLeft  = 2;
static constexpr uint8_t StepSubtract = 4;

#define BLOCK_SIZE 256

__global__ void kernal_set_size_t(size_t * out,
                                  size_t   value,
                                  size_t   count) {
    const size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < count) { out[idx] = value; }
}

__device__ __inline__ int qualifies_as_part_of_run(Range range,
                                                   DEPTH_TYPE required_frontage_height,
                                                   DEPTH_TYPE required_depth) {
    return(range.begin_depth_y <= required_frontage_height) &&
        (range.depth >= required_frontage_height - range.begin_depth_y) &&
        (required_depth <= range.depth - (required_frontage_height - range.begin_depth_y));
}

// Into `dest`, we will write (out_val - run_length)
__global__ void subtract_max_run_right(DEPTH_TYPE depth_required,
                                       Range const * data,
                                       size_t      * dest,
                                       size_t        count,
                                       size_t        max_out_val) {
    size_t out_val = max_out_val;
    size_t cur_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (cur_idx >= count) return;
    Range range = data[cur_idx];
    DEPTH_TYPE const frontage_height = range.begin_depth_y;
    uint8_t          satisfied = 1;
    if (range.depth < depth_required) { goto finish; }
    ++cur_idx; --out_val;
    for ( ; cur_idx < count; ++cur_idx) {
        range = data[cur_idx];
        if ((range.is_begin_run && !satisfied) || (!out_val)) {
            goto finish;
        }
        satisfied = satisfied && (!range.is_begin_run);
        if ((!satisfied) && qualifies_as_part_of_run(range, frontage_height, depth_required)) {
            satisfied = 1;
            --out_val;
        }
    }
 finish:
    dest[threadIdx.x + blockDim.x * blockIdx.x] = out_val;
}

__global__ void subtract_max_run_left_in_place(DEPTH_TYPE depth_required,
                                               Range const * data,
                                               size_t      * dest,
                                               size_t        count) {
    // We don't count the initial one;
    // Similar to the RIGHT version, we subtract, but this time we
    // do that from an initial value.
    size_t cur_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (cur_idx >= count) { return; }
    auto range = data[cur_idx];
    const auto depth_start = range.begin_depth_y;
    if (range.depth < depth_required) {
        return; // don't even need to write anything.
    }
    size_t out_val = dest[cur_idx];
    bool is_last = true;
    bool is_satisfied = true;
    while (cur_idx && out_val) {
        --cur_idx;
        range = data[cur_idx];
        if (is_last && !is_satisfied) { break; }
        is_satisfied = is_satisfied && !is_last;
        if ((!is_satisfied) && qualifies_as_part_of_run(range, depth_start, depth_required)) {
            is_satisfied = 1;
            --out_val;
        }
        is_last = range.is_begin_run;
    }
    dest[threadIdx.x + blockDim.x * blockIdx.x] = out_val;
}

__global__ void subtract_from(size_t amount_to_subtract_from,
                              size_t * data,
                              size_t   count) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < count) {
        data += idx;
        *data = amount_to_subtract_from - *data;
    }
}

size_t ceiling_division(size_t dividend, size_t divisor) {
    return (dividend + divisor - 1) / divisor;
}

int help_runs(DEPTH_TYPE const * begin_depths,
              DEPTH_TYPE const * depths,
              DEPTH_TYPE         required_depth,
              size_t           * output,
              size_t             count,
              uint8_t            steps_todo) {
    int result = 100;
    int cudares = 0;
    Range  * ranges  = NULL;
    Range  * dranges = NULL;
    size_t * dout    = NULL;
    ranges = (Range*) calloc(count, sizeof(Range));
    if (auto local_res = hipMalloc((void**)(&dranges), sizeof(Range) * count)) {
        result=101; goto clean_up;
    }
    if (hipMalloc((void**)(&dout),   sizeof(size_t) * count)) { result=102; goto clean_up; }
    if (!ranges)                                               { result=103; goto clean_up; }
    for (size_t i = 0; i < count; ++i) {
        ranges[i].is_begin_run  = 1;
        ranges[i].begin_depth_y = begin_depths[i];
        ranges[i].depth         = depths[i];
    }
    if (hipMemcpy(dranges, ranges, sizeof(Range) * count, hipMemcpyHostToDevice)) {
        result = 2; goto clean_up;
    }
    if (steps_todo & StepRight) {
        subtract_max_run_right<<<ceiling_division(count, BLOCK_SIZE), BLOCK_SIZE>>>
            (required_depth,
             dranges,
             dout,
             count,
             1024);
    } else {
        kernal_set_size_t<<<ceiling_division(count, BLOCK_SIZE), BLOCK_SIZE>>>
            (dout,
             1024,
             count);
    }
    if (steps_todo & StepLeft) {
        subtract_max_run_left_in_place<<<ceiling_division(count, BLOCK_SIZE), BLOCK_SIZE>>>
            (required_depth,
             dranges,
             dout,
             count);
    }
    if (steps_todo & StepSubtract) {
        subtract_from<<<ceiling_division(count, BLOCK_SIZE), BLOCK_SIZE>>>(1024, dout, count);
    }
    if (hipMemcpy(output, dout, sizeof(size_t) * count, hipMemcpyDeviceToHost)) {
        result = 3; goto clean_up;
    }
    result = 0;
 clean_up:
    free(ranges);
    hipFree(dranges);
    hipFree(dout);
    return result;
}
} // anonymous namespace

int runs_right(DEPTH_TYPE const * begin_depths,
               DEPTH_TYPE const * depths,
               DEPTH_TYPE         required_depth,
               size_t           * output,
               size_t             count) {
    return help_runs(begin_depths,
                     depths,
                     required_depth,
                     output,
                     count,
                     StepRight | StepSubtract);
}

int runs_left(DEPTH_TYPE const * begin_depths,
              DEPTH_TYPE const * depths,
              DEPTH_TYPE         required_depth,
              size_t           * output,
              size_t             count) {
    return help_runs(begin_depths,
                     depths,
                     required_depth,
                     output,
                     count,
                     StepLeft | StepSubtract);
}

int runs_both_sides(DEPTH_TYPE const * begin_depths,
                    DEPTH_TYPE const * depths,
                    DEPTH_TYPE         required_depth,
                    size_t           * output,
                    size_t             count) {
    return help_runs(begin_depths,
                     depths,
                     required_depth,
                     output,
                     count,
                     StepLeft | StepRight | StepSubtract);
}
} // extern "C"

